#include "hip/hip_runtime.h"
/***********************************************************************
 * you are given a partially completed code. Read the code line by line
 * and implement the necessary steps to compute the 1d heat conduction
 * problem given in the handout.
 *
 ***********************************************************************
 *
 * Numerical and analytical solution of the 1D heat conduction problem
 *
 * Author: enter your name
 * Date: enter today's date
 *
 * gcc -O2 -lm -std=c99 1d_heat.c -o heat_1d.exe
 *
 * to execute: ./heat_1d.exe <simulation end time (seconds)>
 *
 *
 */

#include "timer.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#define LX 3.0f /* length of the domain in x-direction  */
#define NX 32   /* includes boundary points on both end */
#define DX LX / ((REAL)(NX - 1))
#define ALPHA 1.0f
#define DT 0.25f * DX *DX / ALPHA
#define RESTRICT __restrict__
#ifndef SINGLE
typedef double REAL;
typedef int    INT;
#define PI M_PI
#else
#define PI M_PI
#endif
#define BLOCKSIZE 256


__global__ void solveHeat_1D(REAL *unew, REAL *u, REAL *x)
{

 int i = blockIdx.x * blockDim.x + threadIdx.x;

    REAL dxi = 1.f / (DX * DX);
    REAL xc, source;

   //  for (i = 1; i < NX - 1; i++) {
   if (i > 0 && i < NX-1 ) {

        xc     = x[ i ];
        source = -(xc * xc - 4.f * xc + 2.f) * exp(-xc); // source term
        unew[ i ]
        = (ALPHA * (u[ i + 1 ] - 2.0f * u[ i ] + u[ i - 1 ]) * dxi + source) * DT + u[ i ];
    }
}

 void exactSolution(REAL *uExact, const REAL *x)
{
         INT i;
    for (i = 0; i < NX; i++) {
 	uExact[ i ] = x[ i ] * x[ i ] * exp(-x[ i ]);
    }
}

void meshGrid(REAL *x)
{
    INT i;
    for (i = 0; i < NX; i++) {
        x[ i ] = DX * (( REAL ) i);
    }
}

void writeOutput(const REAL *x, const REAL *uExact, const REAL *u)
{
    INT   i;
    FILE *output;
    output = fopen("1d_heat.dat", "w");

    for (i = 0; i < NX; i++) {
        fprintf(output, "%10f %10f %10f\n", x[ i ], uExact[ i ], u[ i ]);
    }
    fclose(output);
}

INT main(INT argc, char *argv[])
{
    if (argc < 2) {
        perror("Command-line usage: executableName <end Time (seconds)>");
        exit(1);
    }

    REAL endTime = atof(argv[ 1 ]);

    REAL *uExact, *x;
    REAL *unew, *u, *tmp;

    //  allocate heap memory here for arrays needed in the solution algorithm
    //  read the code carefully to determine those variables

    unew   = hipMallocManaged(&unew,NX*sizeof(REAL));
    u      = hipMallocManaged(&u, NX*sizeof(*u));
    x      = hipMallocManaged(&x,NX*sizeof(*x));

    uExact = (REAL *)calloc(NX, sizeof(*uExact));

    // calculate the x coordinates of each computational point
    meshGrid(x);
    // compute the exact solution to the 1D heat conduction problem
    exactSolution(uExact, x);

    // apply boundary conditions (make sure to apply boundary conditions to both u and unew)
    u[ 0 ]         = 0.f;
    unew[ 0 ]      = 0.f;
    unew[ NX - 1 ] = uExact[ NX - 1 ];
    u[ NX - 1 ]    = uExact[ NX - 1 ];
 int nBlocks = (NX+BLOCKSIZE -1)/BLOCKSIZE ;
    REAL time = 0.f;
    while (time < endTime) {
        // call the solveHeat_1D( ) function here with correct parameters
        // and necessary updates on the solution array
        solveHeat_1D<<<nBlocks,BLOCKSIZE>>>(unew, u, x);

        tmp  = unew;
        unew = u;
        u    = tmp;

        time += DT;
    }

    // call the writeOutput( ) function here with correct parameters

    writeOutput(x, uExact, u);
   
    hipFree(unew);
    hipFree(u);
    hipFree(x);
    free(uExact);
    return EXIT_SUCCESS;
}
